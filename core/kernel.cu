#include "hip/hip_runtime.h"
#pragma once


#include "global_STAP.h"
#include "global_FDAJ.h"

extern int cuda_kernel_grid;
extern int cuda_kernel_block;

///*STAP*/
//#define STAP_CUDA_KERNEL_GRID_NUM 40
//#define STAP_CUDA_KERNEL_BLOCK_NUM 256
///*STAP*/
//
///*FDAJ*/
//#define FDAJ_CUDA_KERNEL_GRID_DIM 40
//#define FDAJ_CUDA_KERNEL_BLOCK_DIM 256//512
//#define FDAJ_CUDA_KERNEL_HANNING_GRID_DIM FDAJ_CUDA_KERNEL_GRID_DIM
//#define FDAJ_CUDA_KERNEL_HANNING_BLOCK_DIM FDAJ_CUDA_KERNEL_BLOCK_DIM
//#define FDAJ_CUDA_KERNEL_MUL_GRID_DIM FDAJ_CUDA_KERNEL_GRID_DIM
//#define FDAJ_CUDA_KERNEL_MUL_BLOCK_DIM FDAJ_CUDA_KERNEL_BLOCK_DIM
//#define FDAJ_CUDA_KERNEL_ABS_GRID_DIM FDAJ_CUDA_KERNEL_GRID_DIM
//#define FDAJ_CUDA_KERNEL_ABS_BLOCK_DIM FDAJ_CUDA_KERNEL_BLOCK_DIM
//#define FDAJ_CUDA_KERNEL_TH_GRID_DIM FDAJ_CUDA_KERNEL_GRID_DIM
//#define FDAJ_CUDA_KERNEL_TH_BLOCK_DIM FDAJ_CUDA_KERNEL_BLOCK_DIM
//#define FDAJ_CUDA_KERNEL_QUANTIZATION_GRID_DIM FDAJ_CUDA_KERNEL_GRID_DIM
//#define FDAJ_CUDA_KERNEL_QUANTIZATION_BLOCK_DIM FDAJ_CUDA_KERNEL_BLOCK_DIM
///*FDAJ*/

__global__ void cuda_kernel_DgemmBatched(SIGNAL_TYPE **dev_array_signal_STAP, SIGNAL_TYPE **dev_array_matrix_R, int col_sig_STAP,
	int lead_dimension_matrix_R)
{
	int i = blockIdx.x;
	const SIGNAL_TYPE one = 1;
	const SIGNAL_TYPE zero = 0;
	hipblasHandle_t dev_blas_handle;
	hipblasCreate(&dev_blas_handle);

	//printf("%d\n", lead_dimension_matrix_R);
	cublasXgemm(dev_blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
		lead_dimension_matrix_R, lead_dimension_matrix_R, col_sig_STAP, &one,
		dev_array_signal_STAP[i], col_sig_STAP,
		dev_array_signal_STAP[i], col_sig_STAP,
		&zero, dev_array_matrix_R[i], lead_dimension_matrix_R);

	hipblasDestroy(dev_blas_handle);
}

__global__ void cuda_kernel_DgemmBatched_scal(SIGNAL_TYPE **dev_array_matrix_R, int col_sig_STAP, int lead_dimension_matrix_R)
{
	int i = blockIdx.x;

	SIGNAL_TYPE scal = 1.0 / col_sig_STAP; //printf("%f\n", scal);
	hipblasHandle_t dev_blas_handle;

	hipblasCreate(&dev_blas_handle);

	cublasXscal(dev_blas_handle, lead_dimension_matrix_R * lead_dimension_matrix_R, &scal, dev_array_matrix_R[i], 1);

	hipblasDestroy(dev_blas_handle);
}

__global__ void cuda_kernel_scal_matrix_R_inver_1col(SIGNAL_TYPE **dev_array_matrix_R_inver_1col, int lead_dimension_matrix_R)
{
	int i = blockIdx.x;
	SIGNAL_TYPE scal = 1.0 / dev_array_matrix_R_inver_1col[i][0];
	//SIGNAL_TYPE scal = __fdiv_rn(1.0f, dev_array_matrix_R_inver_1col[i][0]);
	
	//printf("%f ", scal);

	hipblasHandle_t dev_blas_handle;
	hipblasCreate(&dev_blas_handle);

	cublasXscal(dev_blas_handle, lead_dimension_matrix_R, &scal, dev_array_matrix_R_inver_1col[i], 1);

	hipblasDestroy(dev_blas_handle);

}

__global__ void cuda_kernel_inver_1col_mul(SIGNAL_TYPE **dev_array_matrix_R_inver_1col, SIGNAL_TYPE **dev_array_signal_STAP,
	SIGNAL_TYPE **dev_array_anti_out, int col_sig_STAP, int lead_dimension_matrix_R)
{
	int i = blockIdx.y;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int count = col_sig_STAP;
	SIGNAL_TYPE res = 0;
	SIGNAL_TYPE *current_signal_STAP = dev_array_signal_STAP[i] + tid;

	if (tid < count)
	{
		for (int o = 0; o < lead_dimension_matrix_R; o++)
		{
			res += dev_array_matrix_R_inver_1col[i][o] * (*(current_signal_STAP + o * count));
		}

		dev_array_anti_out[i][tid] = res;
	}
}

__global__ void cuda_kernel_cublasIsamax(SIGNAL_TYPE **dev_input, SIGNAL_TYPE *dev_max, int n)
{
	int i = blockIdx.x;
	int max_index;
	hipblasHandle_t blas_handle;

	hipblasCreate(&blas_handle);

	cublasIXamax(blas_handle, n, (SIGNAL_TYPE *)(dev_input[i]), 1, &max_index);

	dev_max[i] = fabsX(dev_input[i][max_index - 1]);

	hipblasDestroy(blas_handle);


	/*int i = blockIdx.x;
	SIGNAL_TYPE max;
	hipblasHandle_t blas_handle;

	hipblasCreate(&blas_handle);

	hipblasSdot(blas_handle, n, (SIGNAL_TYPE *)(dev_input[i]), 1, (SIGNAL_TYPE *)(dev_input[i]), 1, &max);

	dev_max[i] = sqrtX(max / n) * 2.2910;

	hipblasDestroy(blas_handle);*/
}

__global__ void cuda_kernel_vector_quantization_mul(SIGNAL_TYPE_QUAN **dev_output, SIGNAL_TYPE **dev_input, 
	SIGNAL_TYPE *dev_max, SIGNAL_TYPE pow_quantization, int n, int3 QUAN_config)
{
	int i = blockIdx.y;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int num_all_thread = blockDim.x * gridDim.x;
	SIGNAL_TYPE max = dev_max[i];
	
	if (tid < n)
	{
		SIGNAL_TYPE some_input = dev_input[i][tid];

		SIGNAL_TYPE_QUAN tmp = (SIGNAL_TYPE_QUAN)(QUAN_config.x * roundX(some_input * pow_quantization / max));

		dev_output[i][tid] = (some_input >= 0) ? (tmp + QUAN_config.y) : (tmp + QUAN_config.z);

		/*4bit--1*/
		/*SIGNAL_TYPE_QUAN tmp = (SIGNAL_TYPE_QUAN)(roundX(some_input * pow_quantization / max));

		dev_output[i][tid] = (some_input >= 0) ? (tmp) : (tmp - 1);*/

		/*4bit--2*/
		/*SIGNAL_TYPE_QUAN tmp = (SIGNAL_TYPE_QUAN)(floorX((some_input >= 0 ? some_input : -some_input)* pow_quantization / max));
		dev_output[i][tid] = (some_input >= 0) ? (tmp + 1) : (-tmp - 1);*/

		/*2bit*/
		/*SIGNAL_TYPE_QUAN tmp = (SIGNAL_TYPE_QUAN)(2 * roundX(some_input * pow_quantization / max));

		dev_output[i][tid] = (some_input >= 0) ? (tmp + 1) : (tmp - 1);*/

		/*1bit--1*/
		/*SIGNAL_TYPE_QUAN *tmp = ((SIGNAL_TYPE_QUAN *)(&dev_input[i][tid])) + (sizeof(SIGNAL_TYPE) - 1);

		dev_output[i][tid] = -((*tmp) >> 7);*/

		/*1bit--2*/
		/*SIGNAL_TYPE_QUAN tmp = (SIGNAL_TYPE_QUAN)(roundX(some_input * pow_quantization / max));

		dev_output[i][tid] = (some_input >= 0) ? (tmp) : (tmp + 1);*/
	}

}

//__global__ void cuda_kernel_LCMV_ex(int col_sig_STAP, SIGNAL_TYPE **dev_array_anti_out, SIGNAL_TYPE *dev_anti_out_max,
//	int pow_quantization, SIGNAL_TYPE **__dev_array_signal_STAP, int lead_dimension_matrix_R,
//	int lead_dimension_matrix_A, int size_matrix_R, SIGNAL_TYPE **dev_array_matrix_R,
//	SIGNAL_TYPE **dev_array_matrix_R_inver, SIGNAL_TYPE **dev_array_matrix_R_inver_1col, int *PivotArray,
//	int *infoArray, int q_front, int s, SIGNAL_TYPE_QUAN **STAP_dev_array_anti_out_quan, int3  QUAN_config, 
//	int cuda_kernel_grid, int cuda_kernel_block)
//{
//	hipblasHandle_t matrix_multi_blas_handle;
//	SIGNAL_TYPE ** dev_array_signal_STAP = __dev_array_signal_STAP + q_front * s;
//	hipblasCreate(&matrix_multi_blas_handle);
//
//	//1.
//	//1.1 R=AxA'
//	cuda_kernel_DgemmBatched << <s, 1 >> >(dev_array_signal_STAP, dev_array_matrix_R, col_sig_STAP, lead_dimension_matrix_R);
//
//	//1.2
//	cuda_kernel_DgemmBatched_scal << <s, 1 >> >(dev_array_matrix_R, col_sig_STAP, lead_dimension_matrix_R);
//
//	//2. LU�ֽ������
//	//2.1
//	cublasXgetrfBatched(matrix_multi_blas_handle, lead_dimension_matrix_R, dev_array_matrix_R,
//		lead_dimension_matrix_R, PivotArray, infoArray, s);
//
//	//2.2
//	const SIGNAL_TYPE **temp_dev_array_matrix_R = (const SIGNAL_TYPE **)dev_array_matrix_R;
//	cublasXgetriBatched(matrix_multi_blas_handle, lead_dimension_matrix_R, temp_dev_array_matrix_R,
//		lead_dimension_matrix_R, (const int *)PivotArray, dev_array_matrix_R_inver, lead_dimension_matrix_R,
//		infoArray, s);
//
//	//3.
//	//��matrix_R_inver_1col����С���ɱ�
//	cuda_kernel_scal_matrix_R_inver_1col << <s, 1 >> >(dev_array_matrix_R_inver_1col, lead_dimension_matrix_R);
//
//	///*for (int i = 0; i < 10; i++)
//	//{
//	//	printf("%lf ", dev_array_matrix_R_inver_1col[0][i]);
//	//}*/
//
//	//4.
//	cuda_kernel_inver_1col_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
//		(dev_array_matrix_R_inver_1col, dev_array_signal_STAP, dev_array_anti_out, col_sig_STAP,
//		lead_dimension_matrix_R);
//
//	/*for (int i = 0; i < 10000; i++)
//	{
//		printf("%f ", dev_array_anti_out[0][i]);
//	}
//	printf("\n");*/
//
//	//5.����
//	//5.1
//	cuda_kernel_cublasIsamax << <s, 1 >> >(dev_array_anti_out, dev_anti_out_max, col_sig_STAP);
//
//	/*for (int i = 0; i < s; i++)
//	{
//		printf("%f\n", dev_anti_out_max[i]);
//	}*/
//
//	//5.2
//	cuda_kernel_vector_quantization_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
//		(STAP_dev_array_anti_out_quan, dev_array_anti_out, dev_anti_out_max, pow_quantization, col_sig_STAP, QUAN_config);
//
//
//	hipblasDestroy(matrix_multi_blas_handle);
//
//	/*for (int i = 0; i < 100; i++)
//	{
//		for (int o = 0; o < 4; o++)
//		{
//			for (int j = 0; j < 4; j++)
//			{
//				printf("%f ", dev_array_matrix_R_inver[i][o * 4 + j]);
//			}
//			printf("\n");
//		}
//	}*/
//}

void cuda_kernel_LCMV(int q_front, int s)
{
	hipblasHandle_t matrix_multi_blas_handle;
	SIGNAL_TYPE ** tmp_dev_array_signal_STAP = STAP_dev_array_signal_STAP + q_front * s;
	hipblasCreate(&matrix_multi_blas_handle);

	//1.
	//1.1 R=AxA'
	cuda_kernel_DgemmBatched << <s, 1 >> >(tmp_dev_array_signal_STAP, STAP_dev_array_matrix_R, STAP_col_sig_STAP, STAP_lead_dimension_matrix_R);

	//1.2
	cuda_kernel_DgemmBatched_scal << <s, 1 >> >(STAP_dev_array_matrix_R, STAP_col_sig_STAP, STAP_lead_dimension_matrix_R);

	//2. LU�ֽ������
	//2.1
	cublasXgetrfBatched(matrix_multi_blas_handle, STAP_lead_dimension_matrix_R, STAP_dev_array_matrix_R,
		STAP_lead_dimension_matrix_R, STAP_PivotArray, STAP_infoArray, s);

	//2.2
	const SIGNAL_TYPE **temp_dev_array_matrix_R = (const SIGNAL_TYPE **)STAP_dev_array_matrix_R;
	cublasXgetriBatched(matrix_multi_blas_handle, STAP_lead_dimension_matrix_R, temp_dev_array_matrix_R,
		STAP_lead_dimension_matrix_R, (const int *)STAP_PivotArray, STAP_dev_array_matrix_R_inver, STAP_lead_dimension_matrix_R,
		STAP_infoArray, s);

	//3.
	//��matrix_R_inver_1col����С���ɱ�
	cuda_kernel_scal_matrix_R_inver_1col << <s, 1 >> >(STAP_dev_array_matrix_R_inver_1col, STAP_lead_dimension_matrix_R);

	///*for (int i = 0; i < 10; i++)
	//{
	//	printf("%lf ", dev_array_matrix_R_inver_1col[0][i]);
	//}*/

	//4.
	cuda_kernel_inver_1col_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
		(STAP_dev_array_matrix_R_inver_1col, tmp_dev_array_signal_STAP, STAP_dev_array_anti_out, STAP_col_sig_STAP,
		STAP_lead_dimension_matrix_R);

	/*for (int i = 0; i < 10000; i++)
	{
	printf("%f ", dev_array_anti_out[0][i]);
	}
	printf("\n");*/

	//5.����
	//5.1
	cuda_kernel_cublasIsamax << <s, 1 >> >(STAP_dev_array_anti_out, STAP_dev_anti_out_max, STAP_col_sig_STAP);

	/*for (int i = 0; i < s; i++)
	{
	printf("%f\n", dev_anti_out_max[i]);
	}*/

	//5.2
	cuda_kernel_vector_quantization_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
		(STAP_dev_array_anti_out_quan, STAP_dev_array_anti_out, STAP_dev_anti_out_max, STAP_pow_quantization, STAP_col_sig_STAP, QUAN_config);


	hipblasDestroy(matrix_multi_blas_handle);

	/*for (int i = 0; i < 100; i++)
	{
	for (int o = 0; o < 4; o++)
	{
	for (int j = 0; j < 4; j++)
	{
	printf("%f ", dev_array_matrix_R_inver[i][o * 4 + j]);
	}
	printf("\n");
	}
	}*/
}

__global__ void cuda_kernel_beam_R_inv_v(SIGNAL_TYPE **dev_array_matrix_R_inver, SIGNAL_TYPE
	*dev_beam_vector_real, SIGNAL_TYPE *dev_beam_vector_image, SIGNAL_TYPE **dev_array_R_inv_v_real, 
	SIGNAL_TYPE **dev_array_R_inv_v_image, SIGNAL_TYPE ** dev_array_v_real_R_inv, SIGNAL_TYPE ** 
	dev_array_v_image_R_inv, int lead_dimension_matrix_R)
{
	//�ڼ�������
	int i = blockIdx.x;
	
	//һ������12������ͨ��
	int which_vector_i = lead_dimension_matrix_R * threadIdx.x;
	hipblasHandle_t dev_blas_handle;
	SIGNAL_TYPE one = 1;
	SIGNAL_TYPE minus_one = -1;
	SIGNAL_TYPE zero = 0;
	hipblasCreate(&dev_blas_handle);

	cublasXgemm(dev_blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		lead_dimension_matrix_R, 1, lead_dimension_matrix_R, &one, 
		dev_array_matrix_R_inver[i], lead_dimension_matrix_R, 
		dev_beam_vector_real + which_vector_i, lead_dimension_matrix_R,
		&zero, dev_array_R_inv_v_real[i] + which_vector_i, lead_dimension_matrix_R);

	//if (i == 0 && threadIdx.x == 0)
	//{
	//	for (int o = 0; o < 4; o++)
	//	{
	//		//float ttt = 0;
	//		for (int j = 0; j < 4; j++)
	//		{
	//			printf("%f ", dev_array_matrix_R_inver[0][j * 4 + o]);
	//		}
	//		printf("\n");
	//	}
	//}
	//if (i == 0 && threadIdx.x == 0)
	//{
	//	for (int o = 0; o < 12; o++)
	//	{
	//		//float ttt = 0;
	//		for (int j = 0; j < 4; j++)
	//		{
	//			printf("%f ", dev_beam_vector_real[o * 4 + j]);
	//		}
	//		printf("\n");							
	//	}
	//}
	
	cublasXgemm(dev_blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		1, lead_dimension_matrix_R, lead_dimension_matrix_R, &one, 
		dev_beam_vector_real + which_vector_i, 1,
		dev_array_matrix_R_inver[i], lead_dimension_matrix_R,
		&zero, dev_array_v_real_R_inv[i] + which_vector_i, 1);

	cublasXgemm(dev_blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		lead_dimension_matrix_R, 1, lead_dimension_matrix_R, &one,
		dev_array_matrix_R_inver[i], lead_dimension_matrix_R, 
		dev_beam_vector_image + which_vector_i, lead_dimension_matrix_R,
		&zero, dev_array_R_inv_v_image[i] + which_vector_i, lead_dimension_matrix_R);

	cublasXgemm(dev_blas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
		1, lead_dimension_matrix_R, lead_dimension_matrix_R, &minus_one,
		dev_beam_vector_image + which_vector_i, 1,
		dev_array_matrix_R_inver[i], lead_dimension_matrix_R,
		&zero, dev_array_v_image_R_inv[i] + which_vector_i, lead_dimension_matrix_R);

	hipblasDestroy(dev_blas_handle);

}

__global__ void cuda_kernel_beam_scal_para(SIGNAL_TYPE **dev_array_v_real_R_inv, SIGNAL_TYPE *dev_beam_vector_real, SIGNAL_TYPE **dev_array_v_image_R_inv, SIGNAL_TYPE *dev_beam_vector_image, SIGNAL_TYPE *scal_real, SIGNAL_TYPE *scal_image, int lead_dimension_matrix_R)
{
	int i = blockIdx.x;

	//һ������12������ͨ��
	int which_vector_i = lead_dimension_matrix_R * threadIdx.x;
	hipblasHandle_t dev_blas_handle;
	SIGNAL_TYPE scal_real1, scal_real2, scal_image1, scal_image2;
	SIGNAL_TYPE one = 1;

	hipblasCreate(&dev_blas_handle);

	cublasXdot(dev_blas_handle, lead_dimension_matrix_R, dev_array_v_real_R_inv[i] + which_vector_i, one,
		dev_beam_vector_real + which_vector_i, one, &scal_real1);

	cublasXdot(dev_blas_handle, lead_dimension_matrix_R, dev_array_v_image_R_inv[i] + which_vector_i, one,
		dev_beam_vector_image + which_vector_i, one, &scal_real2);

	cublasXdot(dev_blas_handle, lead_dimension_matrix_R, dev_array_v_real_R_inv[i] + which_vector_i, one,
		dev_beam_vector_image + which_vector_i, one, &scal_image1);

	cublasXdot(dev_blas_handle, lead_dimension_matrix_R, dev_array_v_image_R_inv[i] + which_vector_i, one,
		dev_beam_vector_real + which_vector_i, one, &scal_image2);

	scal_real[i * BEAM_SIG_VECTOR + threadIdx.x] = scal_real1 - scal_real2;
	scal_image[i * BEAM_SIG_VECTOR + threadIdx.x] = scal_image1 + scal_image2;

	//printf("%d %f %f\n", i, scal_real[i], scal_image[i]);

	hipblasDestroy(dev_blas_handle);

}

__global__ void cuda_kernel_beam_scal(SIGNAL_TYPE **dev_array_R_inv_v_real, SIGNAL_TYPE **
	dev_array_R_inv_v_image, SIGNAL_TYPE *scal_real,SIGNAL_TYPE *scal_image, SIGNAL_TYPE **
	dev_array_matrix_R_inver_1col)
{
	int i = blockIdx.x;
	int o = threadIdx.x;
	int j = threadIdx.y;

	//��Լ�ռ䣬��R�������ĵڶ��д��Ȩֵʸ�����鲿
	SIGNAL_TYPE *matrix_R_inver_1col_Sig_d_image = dev_array_matrix_R_inver_1col[i] + blockDim.x;
	
	dev_array_matrix_R_inver_1col[i][o] = 0;
	matrix_R_inver_1col_Sig_d_image[o] = 0;

	__syncthreads();

	SIGNAL_TYPE a = dev_array_R_inv_v_real[i][j * blockDim.x + o];
	SIGNAL_TYPE b = dev_array_R_inv_v_image[i][j * blockDim.x + o];
	SIGNAL_TYPE c = scal_real[i * BEAM_SIG_VECTOR + j];
	SIGNAL_TYPE d = scal_image[i * BEAM_SIG_VECTOR + j];

	atomicAdd(&dev_array_matrix_R_inver_1col[i][o], (a * c + b * d) / (c * c + d * d));
	atomicAdd(&matrix_R_inver_1col_Sig_d_image[o], (b * c - a * d) / (c * c + d * d));

	//printf("%d %d %f\n", i, o, dev_array_matrix_R_inver_1col[i][o]);
	/*printf("%d %f %f\n", i, c, d);*/

}

//__global__ void cuda_kernel_beam_ex(int col_sig_STAP, SIGNAL_TYPE **dev_array_anti_out, SIGNAL_TYPE *dev_anti_out_max,
//	int pow_quantization, SIGNAL_TYPE **dev_array_signal_STAP, int lead_dimension_matrix_R,
//	int lead_dimension_matrix_A, int size_matrix_R, SIGNAL_TYPE **dev_array_matrix_R,
//	SIGNAL_TYPE **dev_array_matrix_R_inver, SIGNAL_TYPE **dev_array_matrix_R_inver_1col, int *PivotArray,
//	int *infoArray, int q_front, int s, SIGNAL_TYPE *dev_beam_vector_real, SIGNAL_TYPE *dev_beam_vector_image,
//	SIGNAL_TYPE **dev_array_R_inv_v_real, SIGNAL_TYPE **dev_array_R_inv_v_image, SIGNAL_TYPE **dev_array_v_real_R_inv,
//	SIGNAL_TYPE **dev_array_v_image_R_inv, SIGNAL_TYPE *dev_beam_scal_real, SIGNAL_TYPE *dev_beam_scal_image,
//	SIGNAL_TYPE_QUAN **STAP_dev_array_anti_out_quan, int3  QUAN_config, int cuda_kernel_grid, int cuda_kernel_block)
//{
//	hipblasHandle_t matrix_multi_blas_handle;
//	dev_array_signal_STAP += q_front * s;
//	hipblasCreate(&matrix_multi_blas_handle);
//
//	const SIGNAL_TYPE one = 1;
//	const SIGNAL_TYPE zero = 0;
//
//	//1.
//	//1.1 R=AxA'
//	cuda_kernel_DgemmBatched << <s, 1 >> >(dev_array_signal_STAP, dev_array_matrix_R, col_sig_STAP, lead_dimension_matrix_R);
//
//	//1.2
//	cuda_kernel_DgemmBatched_scal << <s, 1 >> >(dev_array_matrix_R, col_sig_STAP, lead_dimension_matrix_R);
//
//	//2. LU�ֽ������
//	//2.1
//	cublasXgetrfBatched(matrix_multi_blas_handle, lead_dimension_matrix_R, dev_array_matrix_R,
//		lead_dimension_matrix_R, PivotArray, infoArray, s);
//
//	//2.2
//	const SIGNAL_TYPE **temp_dev_array_matrix_R = (const SIGNAL_TYPE **)dev_array_matrix_R;
//	cublasXgetriBatched(matrix_multi_blas_handle, lead_dimension_matrix_R, temp_dev_array_matrix_R,
//		lead_dimension_matrix_R, (const int *)PivotArray, dev_array_matrix_R_inver, lead_dimension_matrix_R,
//		infoArray, s);
//
//	//3.
//	cuda_kernel_beam_R_inv_v << <s, BEAM_SIG_VECTOR >> >(dev_array_matrix_R_inver, dev_beam_vector_real,
//		dev_beam_vector_image, dev_array_R_inv_v_real, dev_array_R_inv_v_image, dev_array_v_real_R_inv,
//		dev_array_v_image_R_inv, lead_dimension_matrix_R);
//
//	/*SIGNAL_TYPE scal_real[TEST_CHUNK], scal_image[TEST_CHUNK];*/
//	cuda_kernel_beam_scal_para << <s, BEAM_SIG_VECTOR >> >(dev_array_v_real_R_inv, dev_beam_vector_real,
//		dev_array_v_image_R_inv, dev_beam_vector_image, dev_beam_scal_real, dev_beam_scal_image, lead_dimension_matrix_R);
//
//	cuda_kernel_beam_scal << <s, dim3(lead_dimension_matrix_R, BEAM_SIG_VECTOR) >> >(dev_array_R_inv_v_real,
//		dev_array_R_inv_v_image, dev_beam_scal_real, dev_beam_scal_image, dev_array_matrix_R_inver_1col);
//
//	//4.
//	cuda_kernel_inver_1col_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
//		(dev_array_matrix_R_inver_1col, dev_array_signal_STAP, dev_array_anti_out, col_sig_STAP,
//		lead_dimension_matrix_R);
//
//	/*for (int o = 0; o < lead_dimension_matrix_R; o++)
//	{
//	printf("%f ", dev_array_matrix_R_inver[0][o]);
//	}
//	printf("\n");*/
//
//	//5.����
//	//5.1
//	cuda_kernel_cublasIsamax << <s, 1 >> >(dev_array_anti_out, dev_anti_out_max, col_sig_STAP);
//
//	//5.2
//	cuda_kernel_vector_quantization_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
//		(STAP_dev_array_anti_out_quan, dev_array_anti_out, dev_anti_out_max, pow_quantization, col_sig_STAP, QUAN_config);
//
//	hipblasDestroy(matrix_multi_blas_handle);
//}

void cuda_kernel_beam(int q_front, int s)
{
	hipblasHandle_t matrix_multi_blas_handle;
	SIGNAL_TYPE ** tmp_dev_array_signal_STAP = STAP_dev_array_signal_STAP + q_front * s;
	hipblasCreate(&matrix_multi_blas_handle);

	const SIGNAL_TYPE one = 1;
	const SIGNAL_TYPE zero = 0;

	//1.
	//1.1 R=AxA'
	cuda_kernel_DgemmBatched << <s, 1 >> >(tmp_dev_array_signal_STAP, STAP_dev_array_matrix_R, STAP_col_sig_STAP, STAP_lead_dimension_matrix_R);

	//1.2
	cuda_kernel_DgemmBatched_scal << <s, 1 >> >(STAP_dev_array_matrix_R, STAP_col_sig_STAP, STAP_lead_dimension_matrix_R);

	//2. LU�ֽ������
	//2.1
	cublasXgetrfBatched(matrix_multi_blas_handle, STAP_lead_dimension_matrix_R, STAP_dev_array_matrix_R,
		STAP_lead_dimension_matrix_R, STAP_PivotArray, STAP_infoArray, s);

	//2.2
	const SIGNAL_TYPE **temp_dev_array_matrix_R = (const SIGNAL_TYPE **)STAP_dev_array_matrix_R;
	cublasXgetriBatched(matrix_multi_blas_handle, STAP_lead_dimension_matrix_R, temp_dev_array_matrix_R,
		STAP_lead_dimension_matrix_R, (const int *)STAP_PivotArray, STAP_dev_array_matrix_R_inver, STAP_lead_dimension_matrix_R,
		STAP_infoArray, s);

	//3.
	cuda_kernel_beam_R_inv_v << <s, BEAM_SIG_VECTOR >> >(STAP_dev_array_matrix_R_inver, dev_beam_vector_real,
		dev_beam_vector_image, dev_array_R_inv_v_real, dev_array_R_inv_v_image, dev_array_v_real_R_inv,
		dev_array_v_image_R_inv, STAP_lead_dimension_matrix_R);

	/*SIGNAL_TYPE scal_real[TEST_CHUNK], scal_image[TEST_CHUNK];*/
	cuda_kernel_beam_scal_para << <s, BEAM_SIG_VECTOR >> >(dev_array_v_real_R_inv, dev_beam_vector_real,
		dev_array_v_image_R_inv, dev_beam_vector_image, dev_beam_scal_real, dev_beam_scal_image, STAP_lead_dimension_matrix_R);

	cuda_kernel_beam_scal << <s, dim3(STAP_lead_dimension_matrix_R, BEAM_SIG_VECTOR) >> >(dev_array_R_inv_v_real,
		dev_array_R_inv_v_image, dev_beam_scal_real, dev_beam_scal_image, STAP_dev_array_matrix_R_inver_1col);

	//4.
	cuda_kernel_inver_1col_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
		(STAP_dev_array_matrix_R_inver_1col, tmp_dev_array_signal_STAP, STAP_dev_array_anti_out, STAP_col_sig_STAP,
		STAP_lead_dimension_matrix_R);

	/*for (int o = 0; o < lead_dimension_matrix_R; o++)
	{
	printf("%f ", dev_array_matrix_R_inver[0][o]);
	}
	printf("\n");*/

	//5.����
	//5.1
	cuda_kernel_cublasIsamax << <s, 1 >> >(STAP_dev_array_anti_out, STAP_dev_anti_out_max, STAP_col_sig_STAP);

	//5.2
	cuda_kernel_vector_quantization_mul << <dim3(cuda_kernel_grid, s), cuda_kernel_block >> >
		(STAP_dev_array_anti_out_quan, STAP_dev_array_anti_out, STAP_dev_anti_out_max, STAP_pow_quantization, STAP_col_sig_STAP, QUAN_config);

	hipblasDestroy(matrix_multi_blas_handle);
}

__global__ void cuda_kernel_vector_mul(SIGNAL_TYPE **dev_input1, SIGNAL_TYPE *dev_input2,
	SIGNAL_TYPE **dev_result, int n)
{
	int i = blockIdx.y;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int num_all_thread = blockDim.x * gridDim.x;

	if (tid < n)
	{
		dev_result[i][tid] = dev_input1[i][tid] * dev_input2[tid];
	}

}

void cuda_vector_mul(SIGNAL_TYPE **dev_input1, SIGNAL_TYPE *dev_input2, SIGNAL_TYPE **dev_result, int n, int times)
{
	cuda_kernel_vector_mul << <dim3(cuda_kernel_grid, times), cuda_kernel_block >> >
		(dev_input1, dev_input2, dev_result, n);
}

__global__ void cuda_kernel_vector_abs(COMPLEX_TYPE **dev_input, SIGNAL_TYPE **dev_result, int n)
{
	int i = blockIdx.y;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int num_all_thread = blockDim.x * gridDim.x;

	while (tid < n)
	{
		dev_result[i][tid] = sqrtX(dev_input[i][tid].x * dev_input[i][tid].x +
			dev_input[i][tid].y * dev_input[i][tid].y);

		tid += num_all_thread;
	}

}

__global__ void cuda_kernel_cublasSasum(SIGNAL_TYPE **dev_array_s_amp, SIGNAL_TYPE *dev_TH_aver, int n, int T)
{
	int i = blockIdx.x;
	hipblasHandle_t blas_handle;

	hipblasCreate(&blas_handle);

	cublasXasum(blas_handle, n, dev_array_s_amp[i], 1, &dev_TH_aver[i]);

	//HIPFFT_R2C ���ֻ����������ĸ���ϵ������n/2+1����������0���͵�n/2+1���������ظ�
	dev_TH_aver[i] = (dev_TH_aver[i] * 2 - dev_array_s_amp[i][0] - dev_array_s_amp[i][n - 1]) / (2 * n - 2) * T;

	hipblasDestroy(blas_handle);

}

__global__ void cuda_kernel_vector_TH(COMPLEX_TYPE **dev_input1, SIGNAL_TYPE **dev_input2, SIGNAL_TYPE* TH, int n)
{
	int i = blockIdx.y;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int num_all_thread = blockDim.x * gridDim.x;
	COMPLEX_TYPE some_input1;
	SIGNAL_TYPE some_input2;
	SIGNAL_TYPE some_TH = TH[i];

	while (tid < n)
	{
		some_input1 = dev_input1[i][tid];
		some_input2 = dev_input2[i][tid];

		dev_input1[i][tid].x = (some_input2 >= some_TH) ? (some_input1.x * some_TH / some_input2) : (some_input1.x);
		dev_input1[i][tid].y = (some_input2 >= some_TH) ? (some_input1.y * some_TH / some_input2) : (some_input1.y);

		tid += num_all_thread;
	}

}

__global__ void cuda_kernel_TH_filter(int K, COMPLEX_TYPE **dev_array_s_in, SIGNAL_TYPE **dev_array_s_amp,
	SIGNAL_TYPE *dev_TH_aver, int T, int sn, int cycle_number, int cuda_kernel_grid, int cuda_kernel_block)
{
	int n = (sn >> 1) + 1;

	while (K--)
	{
		cuda_kernel_vector_abs << <dim3(cuda_kernel_grid, cycle_number), cuda_kernel_block >> >
			(dev_array_s_in, dev_array_s_amp, n);

		cuda_kernel_cublasSasum << <cycle_number, 1 >> >(dev_array_s_amp, dev_TH_aver, n, T);

		cuda_kernel_vector_TH << <dim3(cuda_kernel_grid, cycle_number), cuda_kernel_block >> >
			(dev_array_s_in, dev_array_s_amp, dev_TH_aver, n);
	}
}

void cuda_TH_filter(int K, COMPLEX_TYPE **dev_array_s_in, SIGNAL_TYPE **dev_array_s_amp,
	SIGNAL_TYPE *dev_TH_aver, int T, int sn, int cycle_number)
{
	cuda_kernel_TH_filter << <1, 1, 0, 0 >> >(K, dev_array_s_in, dev_array_s_amp, dev_TH_aver,
		T, sn, cycle_number, cuda_kernel_grid, cuda_kernel_block);
}

void cuda_vector_quantization(SIGNAL_TYPE **dev_input, SIGNAL_TYPE_QUAN **dev_output, SIGNAL_TYPE pow_quantization, 
	int n, int times)
{
	cuda_kernel_cublasIsamax << <times, 1 >> >(dev_input, FDAJ_dev_s_out_max, n);

	cuda_kernel_vector_quantization_mul << <dim3(cuda_kernel_grid, times),
		cuda_kernel_block >> >(dev_output, dev_input, FDAJ_dev_s_out_max, pow_quantization, n, QUAN_config);

}

__global__ void cuda_kernel_hanning(SIGNAL_TYPE *dev_window, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < n)
	{
		dev_window[tid] = 0.5 * (1 - cosX(2 * PI * tid / (n - 1)));
	}

}

void cuda_hanning(SIGNAL_TYPE *dev_window, int sn)
{
	cuda_kernel_hanning << <cuda_kernel_grid, cuda_kernel_block >> >(dev_window, sn);
}

void cuda_LCMV(int q_front, int s, int sel)
{
	switch (sel)
	{
	case CONFIG_STAP:
		/*cuda_kernel_LCMV_ex << <1, 1 >> >(STAP_col_sig_STAP, STAP_dev_array_anti_out,
			STAP_dev_anti_out_max, STAP_pow_quantization, STAP_dev_array_signal_STAP, STAP_lead_dimension_matrix_R,
			STAP_lead_dimension_matrix_A, STAP_size_matrix_R, STAP_dev_array_matrix_R, STAP_dev_array_matrix_R_inver,
			STAP_dev_array_matrix_R_inver_1col, STAP_PivotArray, STAP_infoArray, q_front, s, STAP_dev_array_anti_out_quan, 
			QUAN_config, cuda_kernel_grid, cuda_kernel_block);*/
		cuda_kernel_LCMV(q_front, s);

		break;
	case CONFIG_BEAM:
		/*cuda_kernel_beam_ex <<<1, 1 >> >(STAP_col_sig_STAP, STAP_dev_array_anti_out,
			STAP_dev_anti_out_max, STAP_pow_quantization, STAP_dev_array_signal_STAP, STAP_lead_dimension_matrix_R,
			STAP_lead_dimension_matrix_A, STAP_size_matrix_R, STAP_dev_array_matrix_R, STAP_dev_array_matrix_R_inver,
			STAP_dev_array_matrix_R_inver_1col, STAP_PivotArray, STAP_infoArray, q_front, s,
			dev_beam_vector_real, dev_beam_vector_image, dev_array_R_inv_v_real, dev_array_R_inv_v_image,
			dev_array_v_real_R_inv, dev_array_v_image_R_inv, dev_beam_scal_real, dev_beam_scal_image, 
			STAP_dev_array_anti_out_quan, QUAN_config, cuda_kernel_grid, cuda_kernel_block);*/
		cuda_kernel_beam(q_front, s);

		break;
	}
	
}